#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <string>
#include <fstream>
#include <cmath>
#include <iomanip>
#include <vector>
#include <complex>
#include <stdio.h>
#include "main.cuh"
#include "readindata.cuh"
#include "emissionfunction.cuh"
#include "Stopwatch.cuh"
#include "arsenal.cuh"
#include "ParameterReader.cuh"
#include "deltafReader.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define AMOUNT_OF_OUTPUT 0    // smaller value means less outputs
//#define THREADS_PER_BLOCK 128  // try optimizing this, also we can define two different threads/block for the separate kernels
//#define FO_CHUNK 6400

using namespace std;

// Class EmissionFunctionArray ------------------------------------------
EmissionFunctionArray::EmissionFunctionArray(ParameterReader* paraRdr_in, Table* chosen_particles_in, Table* pT_tab_in, Table* phi_tab_in, Table* y_tab_in, Table* eta_tab_in, particle_info* particles_in, int Nparticles_in, FO_surf* surf_ptr_in, long FO_length_in, Deltaf_Data * df_data_in)
{
  // kernel parameters
  threadsPerBlock = paraRdr_in->getVal("threads_per_block");
  FO_chunk = paraRdr_in->getVal("chunk_size");

  // control parameters
  OPERATION = paraRdr_in->getVal("operation");
  if(OPERATION == 2) 
  {
    printf("Error: there is no particle sampler in cuda\n");
    exit(-1);
  }
  MODE = paraRdr_in->getVal("mode");
  DF_MODE = paraRdr_in->getVal("df_mode");
  DIMENSION = paraRdr_in->getVal("dimension");
  INCLUDE_BARYON = paraRdr_in->getVal("include_baryon");
  INCLUDE_SHEAR_DELTAF = paraRdr_in->getVal("include_shear_deltaf");
  INCLUDE_BULK_DELTAF = paraRdr_in->getVal("include_bulk_deltaf");
  INCLUDE_BARYONDIFF_DELTAF = paraRdr_in->getVal("include_baryondiff_deltaf");
  OUTFLOW = paraRdr_in->getVal("outflow");
  REGULATE_DELTAF = paraRdr_in->getVal("regulate_deltaf");


  // freezeout surface
  surf_ptr = surf_ptr_in;
  FO_length = FO_length_in;


  // momentum tables
  pT_tab = pT_tab_in;
  phi_tab = phi_tab_in;
  y_tab = y_tab_in;
  eta_tab = eta_tab_in;
  pT_tab_length = pT_tab->getNumberOfRows();
  phi_tab_length = phi_tab->getNumberOfRows();
  y_tab_length = y_tab->getNumberOfRows();
  eta_tab_length = 1;

  if(DIMENSION == 2)
  {
    y_tab_length = 1;
    eta_tab_length = eta_tab->getNumberOfRows();
  }
  y_minus_eta_tab_length = eta_tab->getNumberOfRows();


  // particle info
  particles = particles_in;
  Nparticles = Nparticles_in;
  npart = chosen_particles_in->getNumberOfRows();
  chosen_particles_table = (long*)calloc(npart, sizeof(long));

  for(long ipart = 0; ipart < npart; ipart++)
  {
    long mc_id = chosen_particles_in->get(1, ipart + 1);
    bool found_match = false;

    // search for match in PDG file
    for(long iPDG = 0; iPDG < Nparticles; iPDG++)
    {
      if(particles[iPDG].mc_id == mc_id)
      {
        chosen_particles_table[ipart] = iPDG;
        found_match = true;
        break;
      }
    }
    if(!found_match)
    {
      printf("Chosen particles error: %ld not found in pdg.dat\n", mc_id);
      exit(-1);
    }
  }


  // df coefficients
  df_data = df_data_in;


  // particle spectra of chosen particle species
  momentum_length = pT_tab_length * phi_tab_length * y_tab_length;
  spectra_length = npart * momentum_length;
  if(OPERATION == 1) dN_pTdpTdphidy = (double*)calloc(spectra_length, sizeof(double));


  // spacetime grid
  tau_min = paraRdr_in->getVal("tau_min");
  tau_max = paraRdr_in->getVal("tau_max");
  tau_bins = paraRdr_in->getVal("tau_bins");
  tau_width = (tau_max - tau_min) / (double)tau_bins;

  r_min = paraRdr_in->getVal("r_min");
  r_max = paraRdr_in->getVal("r_max");
  r_bins = paraRdr_in->getVal("r_bins");
  r_width = (r_max - r_min) / (double)r_bins;

  phi_bins = paraRdr_in->getVal("phip_bins");
  phi_width = two_pi / (double)phi_bins;

  double eta_cut = paraRdr_in->getVal("eta_cut");
  eta_min = - eta_cut;
  eta_max = eta_cut;
  eta_bins = paraRdr_in->getVal("eta_bins");
  eta_width = 2.0 * eta_cut / eta_bins;
  if(DIMENSION == 2) 
  {
    eta_bins = 1;
    eta_width = 1.0;
  }


  // spacetime distributions 
  spacetime_length = eta_bins * (tau_bins + r_bins + phi_bins);
  X_length = npart * spacetime_length;
  if(OPERATION == 0) dN_dX = (double*)calloc(X_length, sizeof(double));
}


EmissionFunctionArray::~EmissionFunctionArray()
{

}


__global__ void calculate_dN_pTdpTdphidy_threadReduction(double* dN_pTdpTdphidy_d_blocks, long endFO, long momentum_length, long pT_tab_length, long phi_tab_length, long y_tab_length, long eta_tab_length, double* pT_d, double* trig_d, double* y_d, double* etaValues_d, double* etaWeights_d, double mass_squared, double sign, double prefactor, double baryon, double *T_d, double *tau_d, double *eta_d, double *ux_d, double *uy_d, double *un_d, double *dat_d, double *dax_d, double *day_d, double *dan_d, double *pixx_d, double *pixy_d, double *pixn_d, double *piyy_d, double *piyn_d, double *bulkPi_d, double *alphaB_d, double *Vx_d, double *Vy_d, double *Vn_d, deltaf_coefficients *df_coeff_d, int INCLUDE_BARYON, int REGULATE_DELTAF, int INCLUDE_SHEAR_DELTAF, int INCLUDE_BULK_DELTAF, int INCLUDE_BARYONDIFF_DELTAF, int DIMENSION, int OUTFLOW, int DF_MODE)
{

  // contribution of dN_pTdpTdphidy from each thread
  //long threads = THREADS_PER_BLOCK;
  extern __shared__ double dN_pTdpTdphidy_thread[];

  long icell = (long)threadIdx.x +  (long)blockDim.x * (long)blockIdx.x;  // thread index in kernel
  int ithread = threadIdx.x;                                              // thread index in block

  dN_pTdpTdphidy_thread[ithread] = 0.0;                                   // initialize thread contribution to zero

  __syncthreads();                                                        // sync threads for each block

  if(icell < endFO)
  {
    double tau = tau_d[icell];        // longitudinal proper time
    double tau2 = tau * tau;

    if(DIMENSION == 3)
    {
      etaValues_d[0] = eta_d[icell];  // spacetime rapidity from surface
    }
    double dat = dat_d[icell];        // dsigma_mu
    double dax = dax_d[icell];
    double day = day_d[icell];
    double dan = dan_d[icell];

    double ux = ux_d[icell];          // u^mu
    double uy = uy_d[icell];          // enforce u.u = 1
    double un = un_d[icell];
    double ux2 = ux * ux;             // useful expressions
    double uy2 = uy * uy;
    double utperp = sqrt(1.0 + ux2 + uy2);
    double tau2_un = tau2 * un;
    double ut = sqrt(utperp * utperp  +  tau2_un * un);
    double ut2 = ut * ut;

    bool positive_time_volume = (ut * dat  +  ux * dax  +  uy * day  +  un * dan > 0.0);

    double T = T_d[icell];            // temperature

    double pitt = 0.0;                // pi^munu
    double pitx = 0.0;                // enforce pi.u = 0, Tr(pi) = 0
    double pity = 0.0;
    double pitn = 0.0;
    double pixx = 0.0;
    double pixy = 0.0;
    double pixn = 0.0;
    double piyy = 0.0;
    double piyn = 0.0;
    double pinn = 0.0;

    if(INCLUDE_SHEAR_DELTAF)
    {
      pixx = pixx_d[icell];
      pixy = pixy_d[icell];
      pixn = pixn_d[icell];
      piyy = piyy_d[icell];
      piyn = piyn_d[icell];
      pinn = (pixx * (ux2 - ut2)  +  piyy * (uy2 - ut2)  +  2.0 * (pixy * ux * uy  +  tau2_un * (pixn * ux  +  piyn * uy))) / (tau2 * utperp * utperp);
      pitn = (pixn * ux  +  piyn * uy  +  tau2_un * pinn) / ut;
      pity = (pixy * ux  +  piyy * uy  +  tau2_un * piyn) / ut;
      pitx = (pixx * ux  +  pixy * uy  +  tau2_un * pixn) / ut;
      pitt = (pitx * ux  +  pity * uy  +  tau2_un * pitn) / ut;
    }

    double bulkPi = 0.0;              // bulk pressure

    if(INCLUDE_BULK_DELTAF)
    {
      bulkPi = bulkPi_d[icell];
    }

    double alphaB = 0.0;              // muB / T
    double Vt = 0.0;                  // V^mu
    double Vx = 0.0;                  // enforce orthogonality V.u = 0
    double Vy = 0.0;
    double Vn = 0.0;
    double chem = 0.0;

    if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
    {
      alphaB = alphaB_d[icell];
      chem = baryon * alphaB;
      Vx = Vx_d[icell];
      Vy = Vy_d[icell];
      Vn = Vn_d[icell];
      Vt = (Vx * ux  +  Vy * uy  +  Vn * tau2_un) / ut;
    }

    double tau2_pitn = tau2 * pitn;   // useful expressions
    double tau2_pixn = tau2 * pixn;
    double tau2_piyn = tau2 * piyn;
    double tau4_pinn = tau2 * tau2 * pinn;
    double tau2_Vn = tau2 * Vn;

    // get df coefficients
    double c0 = df_coeff_d->c0;       // 14 moment coefficients
    double c1 = df_coeff_d->c1;
    double c2 = df_coeff_d->c2;
    double c3 = df_coeff_d->c3;
    double c4 = df_coeff_d->c4;
    double shear14_coeff = df_coeff_d->shear14_coeff;
    

    double F = df_coeff_d->F;         // Chapman Enskog
    double G = df_coeff_d->G;
    double betabulk = df_coeff_d->betabulk;
    double betaV = df_coeff_d->betaV;
    double betapi = df_coeff_d->betapi;
    double baryon_enthalpy_ratio = df_coeff_d->baryon_enthalpy_ratio;

    // shear and bulk coefficients
    double shear_coeff = 0.0;
    double bulk0_coeff = 0.0;
    double bulk1_coeff = 0.0;
    double bulk2_coeff = 0.0;
    double diff0_coeff = 0.0;
    double diff1_coeff = 0.0;

    switch(DF_MODE)
    {
      case 1: // 14 moment
      {
        shear_coeff = 1.0 / shear14_coeff;
        bulk0_coeff = (c0 - c2) * mass_squared * bulkPi;
        bulk1_coeff = c1 * baryon * bulkPi;
        bulk2_coeff = (4.*c2 - c0) * bulkPi;
        diff0_coeff = c3 * baryon;
        diff1_coeff = c4;
        break;
      }
      case 2: // Chapman enskog
      {
        shear_coeff = 0.5 / (betapi * T);
        bulk0_coeff = F / (T * T * betabulk) * bulkPi;
        bulk1_coeff = G / betabulk * baryon * bulkPi;
        bulk2_coeff = bulkPi / (3.0 * T * betabulk);
        diff0_coeff = baryon_enthalpy_ratio / betaV;
        diff1_coeff = baryon / betaV;
        break;
      }
      default:
      {
        printf("Error: set df_mode = (1,2) in parameters.dat\n");
      }
    }


    // loop over momentum
    for(long ipT = 0; ipT < pT_tab_length; ipT++)
    {
      long iP1D = phi_tab_length * ipT;

      double pT = pT_d[ipT];

      double mT = sqrt(mass_squared  +  pT * pT);
      double mT_over_tau = mT / tau;

      for(long iphip = 0; iphip < phi_tab_length; iphip++)
      {
        long iP2D = y_tab_length * (iphip + iP1D);

        double px = pT * trig_d[iphip];
        double py = pT * trig_d[iphip + phi_tab_length];

        double px_dax = px * dax;   // useful expressions
        double py_day = py * day;

        double px_ux = px * ux;
        double py_uy = py * uy;

        double pixx_px_px = pixx * px * px;
        double piyy_py_py = piyy * py * py;
        double pitx_px = pitx * px;
        double pity_py = pity * py;
        double pixy_px_py = pixy * px * py;
        double tau2_pixn_px = tau2_pixn * px;
        double tau2_piyn_py = tau2_piyn * py;

        double Vx_px = Vx * px;
        double Vy_py = Vy * py;

        for(long iy = 0; iy < y_tab_length; iy++)
        {
          // momentum_length index
          long iP3D = iy + iP2D;

          double y = y_d[iy];

          //long iP = ipT  +  pT_tab_length * (iphip  +  phi_tab_length * iy);

          if(positive_time_volume)
          {
            double eta_integral = 0.0;

            // sum over eta
            for(long ieta = 0; ieta < eta_tab_length; ieta++)
            {
              double eta = etaValues_d[ieta];
              double eta_weight = etaWeights_d[ieta];

              double sinhyeta = sinh(y - eta);
              double coshyeta = sqrt(1.0  +  sinhyeta * sinhyeta);

              double pt = mT * coshyeta;           // p^tau
              double pn = mT_over_tau * sinhyeta;  // p^eta

              double pdotdsigma = pt * dat  +  px_dax  +  py_day  +  pn * dan;

              if(OUTFLOW && pdotdsigma <= 0.0) continue;  // enforce outflow

              double E = pt * ut  -  px_ux  -  py_uy  -  pn * tau2_un;  // u.p
              double feq = 1.0 / (exp(E/T  -  chem) + sign);

              double feqbar = 1.0  -  sign * feq;

              // pi^munu.p_mu.p_nu
              double pimunu_pmu_pnu = pitt * pt * pt  +  pixx_px_px  +  piyy_py_py  +  tau4_pinn * pn * pn
                  + 2.0 * (-(pitx_px + pity_py) * pt  +  pixy_px_py  +  pn * (tau2_pixn_px  +  tau2_piyn_py  -  tau2_pitn * pt));

              // V^mu.p_mu
              double Vmu_pmu = Vt * pt  -  Vx_px  -  Vy_py  -  tau2_Vn * pn;

              double df;

              switch(DF_MODE)
              {
                case 1: // 14 moment
                {
                  double df_shear = shear_coeff * pimunu_pmu_pnu;
                  double df_bulk = bulk0_coeff  +  (bulk1_coeff  +  bulk2_coeff * E) * E;
                  double df_diff = (diff0_coeff  +  diff1_coeff * E) * Vmu_pmu;

                  df = feqbar * (df_shear + df_bulk + df_diff);
                  break;
                }
                case 2: // Chapman enskog
                {
                  double df_shear = shear_coeff * pimunu_pmu_pnu / E;
                  double df_bulk = bulk0_coeff * E  +  bulk1_coeff  +  bulk2_coeff * (E  -  mass_squared / E);
                  double df_diff = (diff0_coeff  -  diff1_coeff / E) * Vmu_pmu;

                  df = feqbar * (df_shear + df_bulk + df_diff);
                  break;
                }
                default:
                {
                  printf("Error: set df_mode = (1,2) in parameters.dat\n");
                }
              } // DF_MODE

              if(REGULATE_DELTAF) df = max(-1.0, min(df, 1.0));

              double f = feq * (1.0 + df);

              eta_integral += eta_weight * pdotdsigma * f;

            } // ieta

            dN_pTdpTdphidy_thread[ithread] = prefactor * eta_integral;
          }
          else
          {
            dN_pTdpTdphidy_thread[ithread] = 0.0;
          }

          // perform reduction over threads in each block:
          int N = blockDim.x;  // number of threads in block (must be power of 2)
          __syncthreads();     // prepare threads for reduction

          while(N != 1)
          {
            N /= 2;

            if(ithread < N) // reduce thread pairs
            {
              dN_pTdpTdphidy_thread[ithread] += dN_pTdpTdphidy_thread[ithread + N];
            }
            __syncthreads();
          }

          // store block's contribution to the spectra
          if(ithread == 0)
          {
            long iP_block = iP3D  +  blockIdx.x * momentum_length;

            dN_pTdpTdphidy_d_blocks[iP_block] = dN_pTdpTdphidy_thread[0];
          }

        } // iy

      } // iphip

    } // ipT

  } // icell < endFO

} // end function


__global__ void calculate_dN_dX_threadReduction(double *dN_dX_d_blocks, long endFO, long tau_bins, long r_bins, long phi_bins, long eta_bins, double tau_min, double r_min, double eta_min, double tau_width, double r_width, double phi_width, double eta_width, long pT_tab_length, long phi_tab_length, long y_minus_eta_tab_length, double *pT_d, double *pT_weight_d, double *trig_d, double *phip_weight_d, double *y_minus_eta_d, double *y_minus_eta_weight_d, double mass_squared, double sign, double prefactor, double baryon, double *T_d, double *tau_d, double *x_d, double *y_d, double *eta_d, double *ux_d, double *uy_d, double *un_d, double *dat_d, double *dax_d, double *day_d, double *dan_d, double *pixx_d, double *pixy_d, double *pixn_d, double *piyy_d, double *piyn_d, double *bulkPi_d, double *alphaB_d, double *Vx_d, double *Vy_d, double *Vn_d, deltaf_coefficients *df_coeff_d, int INCLUDE_BARYON, int REGULATE_DELTAF, int INCLUDE_SHEAR_DELTAF, int INCLUDE_BULK_DELTAF, int INCLUDE_BARYONDIFF_DELTAF, int DIMENSION, int OUTFLOW, int DF_MODE)
{
  // contribution of dN_pTdpTdphidy from each thread
  //__shared__ double dN_pTdpTdphidy_thread[THREADS_PER_BLOCK];
  long icell = (long)threadIdx.x +  (long)blockDim.x * (long)blockIdx.x;  // thread index in kernel
  long ithread = (long)threadIdx.x;                                       // thread index in block
  long N = blockDim.x;                                                    // number threads per block

  long spacetime_block = (long)blockIdx.x * eta_bins * (tau_bins + r_bins + phi_bins);

  /*
  extern __shared__ double dN_taudtaudeta[];  // worried about going over stack
  
  if(ithread == 0)
  {
    for(long i = 0; i < spacetime_length; i++)
    {
      dN_dX[i] = 0.0;
    }
  }
  */

  __syncthreads();                                             

  if(icell < endFO)
  {
    double tau = tau_d[icell];        // longitudinal proper time
    double tau2 = tau * tau;

    double x = x_d[icell];
    double y = y_d[icell];

    double r = sqrt(x*x + y*y);
    double phi = atan2(y, x);
    if(phi < 0.0) phi += two_pi;

    double eta = 0.0;
    if(DIMENSION == 3)
    {
      eta = eta_d[icell];             // spacetime rapidity from surface
    }

    double dat = dat_d[icell];        // dsigma_mu
    double dax = dax_d[icell];
    double day = day_d[icell];
    double dan = dan_d[icell];

    double ux = ux_d[icell];          // u^mu
    double uy = uy_d[icell];          // enforce u.u = 1
    double un = un_d[icell];
    double ux2 = ux * ux;             // useful expressions
    double uy2 = uy * uy;
    double utperp = sqrt(1.0 + ux2 + uy2);
    double tau2_un = tau2 * un;
    double ut = sqrt(utperp * utperp  +  tau2_un * un);
    double ut2 = ut * ut;

    bool positive_time_volume = (ut * dat  +  ux * dax  +  uy * day  +  un * dan) > 0.0;

    double T = T_d[icell];            // temperature

    double pitt = 0.0;                // pi^munu
    double pitx = 0.0;                // enforce pi.u = 0, Tr(pi) = 0
    double pity = 0.0;
    double pitn = 0.0;
    double pixx = 0.0;
    double pixy = 0.0;
    double pixn = 0.0;
    double piyy = 0.0;
    double piyn = 0.0;
    double pinn = 0.0;

    if(INCLUDE_SHEAR_DELTAF)
    {
      pixx = pixx_d[icell];
      pixy = pixy_d[icell];
      pixn = pixn_d[icell];
      piyy = piyy_d[icell];
      piyn = piyn_d[icell];
      pinn = (pixx * (ux2 - ut2)  +  piyy * (uy2 - ut2)  +  2.0 * (pixy * ux * uy  +  tau2_un * (pixn * ux  +  piyn * uy))) / (tau2 * utperp * utperp);
      pitn = (pixn * ux  +  piyn * uy  +  tau2_un * pinn) / ut;
      pity = (pixy * ux  +  piyy * uy  +  tau2_un * piyn) / ut;
      pitx = (pixx * ux  +  pixy * uy  +  tau2_un * pixn) / ut;
      pitt = (pitx * ux  +  pity * uy  +  tau2_un * pitn) / ut;
    }

    double bulkPi = 0.0;              // bulk pressure

    if(INCLUDE_BULK_DELTAF)
    {
      bulkPi = bulkPi_d[icell];
    }

    double alphaB = 0.0;              // muB / T
    double Vt = 0.0;                  // V^mu
    double Vx = 0.0;                  // enforce orthogonality V.u = 0
    double Vy = 0.0;
    double Vn = 0.0;
    double chem = 0.0;

    if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
    {
      alphaB = alphaB_d[icell];
      chem = baryon * alphaB;
      Vx = Vx_d[icell];
      Vy = Vy_d[icell];
      Vn = Vn_d[icell];
      Vt = (Vx * ux  +  Vy * uy  +  Vn * tau2_un) / ut;
    }

    double tau2_pitn = tau2 * pitn;   // useful expressions
    double tau2_pixn = tau2 * pixn;
    double tau2_piyn = tau2 * piyn;
    double tau4_pinn = tau2 * tau2 * pinn;
    double tau2_Vn = tau2 * Vn;

    // get df coefficients
    double c0 = df_coeff_d->c0;       // 14 moment coefficients
    double c1 = df_coeff_d->c1;
    double c2 = df_coeff_d->c2;
    double c3 = df_coeff_d->c3;
    double c4 = df_coeff_d->c4;
    double shear14_coeff = df_coeff_d->shear14_coeff;
    
    double F = df_coeff_d->F;         // Chapman Enskog
    double G = df_coeff_d->G;
    double betabulk = df_coeff_d->betabulk;
    double betaV = df_coeff_d->betaV;
    double betapi = df_coeff_d->betapi;
    double baryon_enthalpy_ratio = df_coeff_d->baryon_enthalpy_ratio;

    // shear and bulk coefficients
    double shear_coeff = 0.0;
    double bulk0_coeff = 0.0;
    double bulk1_coeff = 0.0;
    double bulk2_coeff = 0.0;
    double diff0_coeff = 0.0;
    double diff1_coeff = 0.0;

    switch(DF_MODE)
    {
      case 1: // 14 moment
      {
        shear_coeff = 1.0 / shear14_coeff;
        bulk0_coeff = (c0 - c2) * mass_squared * bulkPi;
        bulk1_coeff = c1 * baryon * bulkPi;
        bulk2_coeff = (4.*c2 - c0) * bulkPi;
        diff0_coeff = c3 * baryon;
        diff1_coeff = c4;
        break;
      }
      case 2: // Chapman enskog
      {
        shear_coeff = 0.5 / (betapi * T);
        bulk0_coeff = F / (T * T * betabulk) * bulkPi;
        bulk1_coeff = G / betabulk * baryon * bulkPi;
        bulk2_coeff = bulkPi / (3.0 * T * betabulk);
        diff0_coeff = baryon_enthalpy_ratio / betaV;
        diff1_coeff = baryon / betaV;
        break;
      }
      default:
      {
        printf("Error: set df_mode = (1,2) in parameters.dat\n");
      }
    }

    double dN_deta = 0.0;

    if(positive_time_volume)
    {
      // loop over momentum
      for(long ipT = 0; ipT < pT_tab_length; ipT++)
      {
        double pT = pT_d[ipT];
        double pT_weight = pT_weight_d[ipT];

        double mT = sqrt(mass_squared  +  pT * pT);
        double mT_over_tau = mT / tau;

        for(long iphip = 0; iphip < phi_tab_length; iphip++)
        {
          double phip_weight = phip_weight_d[iphip];

          double px = pT * trig_d[iphip];
          double py = pT * trig_d[iphip + phi_tab_length];

          double px_dax = px * dax;   // useful expressions
          double py_day = py * day;

          double px_ux = px * ux;
          double py_uy = py * uy;

          double pixx_px_px = pixx * px * px;
          double piyy_py_py = piyy * py * py;
          double pitx_px = pitx * px;
          double pity_py = pity * py;
          double pixy_px_py = pixy * px * py;
          double tau2_pixn_px = tau2_pixn * px;
          double tau2_piyn_py = tau2_piyn * py;

          double Vx_px = Vx * px;
          double Vy_py = Vy * py;

          // integral over y (centered around eta point) 
          for(long iyeta = 0; iyeta < y_minus_eta_tab_length; iyeta++)     
          {
            double y_minus_eta = y_minus_eta_d[iyeta];                 // this should be a seperate table for spacetime integration (borrow from eta gauss table) 
            double y_minus_eta_weight = y_minus_eta_weight_d[iyeta];
          
            double sinhyeta = sinh(y_minus_eta);
            double coshyeta = sqrt(1.0  +  sinhyeta * sinhyeta);

            double pt = mT * coshyeta;           // p^tau
            double pn = mT_over_tau * sinhyeta;  // p^eta

            double pdotdsigma = pt * dat  +  px_dax  +  py_day  +  pn * dan;

            if(OUTFLOW && pdotdsigma <= 0.0) continue;  // enforce outflow

            double E = pt * ut  -  px_ux  -  py_uy  -  pn * tau2_un;  // u.p
            double feq = 1.0 / (exp(E/T  -  chem) + sign);

            double feqbar = 1.0  -  sign * feq;

            // pi^munu.p_mu.p_nu
            double pimunu_pmu_pnu = pitt * pt * pt  +  pixx_px_px  +  piyy_py_py  +  tau4_pinn * pn * pn
                + 2.0 * (-(pitx_px + pity_py) * pt  +  pixy_px_py  +  pn * (tau2_pixn_px  +  tau2_piyn_py  -  tau2_pitn * pt));

            // V^mu.p_mu
            double Vmu_pmu = Vt * pt  -  Vx_px  -  Vy_py  -  tau2_Vn * pn;

            double df;

            switch(DF_MODE)
            {
              case 1: // 14 moment
              {
                double df_shear = shear_coeff * pimunu_pmu_pnu;
                double df_bulk = bulk0_coeff  +  (bulk1_coeff  +  bulk2_coeff * E) * E;
                double df_diff = (diff0_coeff  +  diff1_coeff * E) * Vmu_pmu;

                df = feqbar * (df_shear + df_bulk + df_diff);
                break;
              }
              case 2: // Chapman enskog
              {
                double df_shear = shear_coeff * pimunu_pmu_pnu / E;
                double df_bulk = bulk0_coeff * E  +  bulk1_coeff  +  bulk2_coeff * (E  -  mass_squared / E);
                double df_diff = (diff0_coeff  -  diff1_coeff / E) * Vmu_pmu;

                df = feqbar * (df_shear + df_bulk + df_diff);
                break;
              }
              default:
              {
                printf("Error: set df_mode = (1,2) in parameters.dat\n");
              }
            } // DF_MODE

            if(REGULATE_DELTAF) df = max(-1.0, min(df, 1.0));

            double f = feq * (1.0 + df);

            dN_deta += prefactor * pT_weight * phip_weight * y_minus_eta_weight * pdotdsigma * f;

          } // iy

        } // iphip

      } // ipT

    } // if positive time volume


    // bin the spacetime distributions one thread at a time (try N = 32)
    __syncthreads();     

    for(long n = 0; n < N; n++)
    {
      if(ithread == n && icell < endFO)
      {
        long ieta = 0;
        if(DIMENSION == 3)
        {
          ieta = (long)floor((eta - eta_min) / eta_width); 
        }
        
        if(ieta >= 0 && ieta < eta_bins)
        {
          long itau = (long)floor((tau - tau_min) / tau_width); 
          long ir = (long)floor((r - r_min) / r_width);
          long iphi = (long)floor(phi / phi_width);

          if(itau >= 0 && itau < tau_bins) 
          {
            dN_dX_d_blocks[ieta  +  eta_bins * itau  +  spacetime_block] += dN_deta;
          }

          if(ir >= 0 && ir < r_bins) 
          {
            dN_dX_d_blocks[ieta  +  eta_bins * (ir + tau_bins)  +  spacetime_block] += dN_deta;
          }

          if(iphi >= 0 && iphi < phi_bins) 
          {
            dN_dX_d_blocks[ieta  +  eta_bins * (iphi + tau_bins + r_bins) +  spacetime_block] += dN_deta;
          }
        }
      }
      __syncthreads();
    }
   
  } // icell < endFO

} // end function



/*


__global__ void calculate_dN_pTdpTdphidy_feqmod_threadReduction(double* dN_pTdpTdphidy_d_blocks, long endFO, long momentum_length, long pT_tab_length, long phi_tab_length, long y_tab_length, long eta_tab_length, double* pT_d, double* trig_d, double* y_d, double* etaValues_d, double* etaWeights_d, double mass_squared, double sign, double prefactor, double baryon, double *T_d, double *tau_d, double *eta_d, double *ux_d, double *uy_d, double *un_d, double *dat_d, double *dax_d, double *day_d, double *dan_d, double *pixx_d, double *pixy_d, double *pixn_d, double *piyy_d, double *piyn_d, double *bulkPi_d, double *alphaB_d, double *Vx_d, double *Vy_d, double *Vn_d, deltaf_coefficients *df_coeff_d, int INCLUDE_BARYON, int REGULATE_DELTAF, int INCLUDE_SHEAR_DELTAF, int INCLUDE_BULK_DELTAF, int INCLUDE_BARYONDIFF_DELTAF, int DIMENSION, int OUTFLOW, int DF_MODE, double DETA_MIN, double MASS_PION0, Gauss_Laguerre * laguerre)
{

  // contribution of dN_pTdpTdphidy from each thread
  __shared__ double dN_pTdpTdphidy_thread[THREADS_PER_BLOCK];

  long icell = (long)threadIdx.x +  (long)blockDim.x * (long)blockIdx.x;  // thread index in kernel
  int ithread = threadIdx.x;                                              // thread index in block

  dN_pTdpTdphidy_thread[ithread] = 0.0;                                   // initialize thread contribution to zero

  __syncthreads();                                                        // sync threads for each block

  if(icell < endFO)
  {
    /// gauss laguerre roots
    const int pbar_pts = laguerre->points;

    double * pbar_root1 = laguerre->root[1];
    double * pbar_root2 = laguerre->root[2];

    double * pbar_weight1 = laguerre->weight[1];
    double * pbar_weight2 = laguerre->weight[2];

    double A[3][3];

    double tau = tau_d[icell];        // longitudinal proper time
    double tau2 = tau * tau;

    if(DIMENSION == 3)
    {
      etaValues_d[0] = eta_d[icell];  // spacetime rapidity from surface
    }
    double dat = dat_d[icell];        // dsigma_mu
    double dax = dax_d[icell];
    double day = day_d[icell];
    double dan = dan_d[icell];

    double ux = ux_d[icell];          // u^mu
    double uy = uy_d[icell];          // enforce u.u = 1
    double un = un_d[icell];
    double ux2 = ux * ux;             // useful expressions
    double uy2 = uy * uy;
    double uperp == sqrt(ux2 + uy2);
    double utperp = sqrt(1.0 + ux2 + uy2);
    double tau2_un = tau2 * un;
    double ut = sqrt(utperp * utperp  +  tau2_un * un);
    double ut2 = ut * ut;

    bool positive_time_volume = (ut * dat  +  ux * dax  +  uy * day  +  un * dan > 0.0);

    double T = T_d[icell];            // temperature

    double pitt = 0.0;                // pi^munu
    double pitx = 0.0;                // enforce pi.u = 0, Tr(pi) = 0
    double pity = 0.0;
    double pitn = 0.0;
    double pixx = 0.0;
    double pixy = 0.0;
    double pixn = 0.0;
    double piyy = 0.0;
    double piyn = 0.0;
    double pinn = 0.0;

    if(INCLUDE_SHEAR_DELTAF)
    {
      pixx = pixx_d[icell];
      pixy = pixy_d[icell];
      pixn = pixn_d[icell];
      piyy = piyy_d[icell];
      piyn = piyn_d[icell];
      pinn = (pixx * (ux2 - ut2)  +  piyy * (uy2 - ut2)  +  2.0 * (pixy * ux * uy  +  tau2_un * (pixn * ux  +  piyn * uy))) / (tau2 * utperp * utperp);
      pitn = (pixn * ux  +  piyn * uy  +  tau2_un * pinn) / ut;
      pity = (pixy * ux  +  piyy * uy  +  tau2_un * piyn) / ut;
      pitx = (pixx * ux  +  pixy * uy  +  tau2_un * pixn) / ut;
      pitt = (pitx * ux  +  pity * uy  +  tau2_un * pitn) / ut;
    }

    double bulkPi = 0.0;              // bulk pressure

    if(INCLUDE_BULK_DELTAF)
    {
      bulkPi = bulkPi_d[icell];
    }

    double alphaB = 0.0;              // muB / T
    double Vt = 0.0;                  // V^mu
    double Vx = 0.0;                  // enforce orthogonality V.u = 0
    double Vy = 0.0;
    double Vn = 0.0;
    double chem = 0.0;

    if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
    {
      alphaB = alphaB_d[icell];
      chem = baryon * alphaB;
      Vx = Vx_d[icell];
      Vy = Vy_d[icell];
      Vn = Vn_d[icell];
      Vt = (Vx * ux  +  Vy * uy  +  Vn * tau2_un) / ut;
    }

    // need basis vector components
    double sinhL = tau * un / utperp;
    double coshL = ut / utperp;

    double Xt = uperp * coshL;
    double Xn = uperp * sinhL / tau;
    double Zt = sinhL;
    double Zn = coshL / tau;

    double tau2_Xn = tau2 * Xn;
    double tau2_Zn = tau2 * Zn;

    double Xx, Xy;
    double Yx, Yy;

    if(uperp < 1.e-5)
    {
      // stops (ux=0)/(uperp=0) nans for freezeout cells with no transverse flow
      Xx = 1.0;   Yx = 0.0;
      Xy = 0.0;   Yy = 1.0;
    }
    else
    {
        Xx = utperp * ux / uperp;   Yx = - uy / uperp;
        Xy = utperp * uy / uperp;   Yy = ux / uperp;
    }

    // pimunu LRF components
    double pixx_LRF = pitt*Xt*Xt + pixx*Xx*Xx + piyy*Xy*Xy + tau2*tau2*pinn*Xn*Xn
            + 2.0 * (-Xt*(pitx*Xx + pity*Xy) + pixy*Xx*Xy + tau2*Xn*(pixn*Xx + piyn*Xy - pitn*Xt));

    double pixy_LRF = Yx*(-pitx*Xt + pixx*Xx + pixy*Xy + tau2*pixn*Xn) + Yy*(-pity*Xt + pixy*Xx + piyy*Xy + tau2*piyn*Xn);

    double pixz_LRF = Zt*(pitt*Xt - pitx*Xx - pity*Xy - tau2*pitn*Xn) - tau2*Zn*(pitn*Xt - pixn*Xx - piyn*Xy - tau2*pinn*Xn);

    double piyy_LRF = pixx*Yx*Yx + 2.0*pixy*Yx*Yy + piyy*Yy*Yy;

    double piyz_LRF = -Zt*(pitx*Yx + pity*Yy) + tau2*Zn*(pixn*Yx + piyn*Yy);

    double pizz_LRF = - (pixx_LRF + piyy_LRF);

    // get df coefficients
    double F = df_coeff_d->F;
    double G = df_coeff_d->G;
    double betabulk = df_coeff_d->betabulk;
    double baryon_enthalpy_ratio = df_coeff_d->baryon_enthalpy_ratio;
    double betaV = df_coeff_d->betaV;
    double betapi = df_coeff_d->betapi;

    double z = df_coeff_d->z;
    double lambda = df_coeff_d->lambda;
    double delta_z = df_coeff_d->delta_z;
    double delta_lambda = df_coeff_d->delta_lambda;


    double shear_mod = 0.5 / betapi;
    double bulk_mod, T_mod, chem_mod;

    // linearized shear and bulk coefficients
    double shear_coeff = 0.5 / (betapi * T);
    double bulk0_coeff, bulk1_coeff, bulk2_coeff;

    switch(DF_MODE)
    {
      case 3: // Mike
      {
        bulk_mod = bulkPi / (3.0 * betabulk);
        T_mod = T  +  F * bulkPi / betabulk;
        chem_mod = baryon * (alphaB  + G * bulkPi / betabulk);

        bulk0_coeff = F / (T * T * betabulk) * bulkPi;
        bulk1_coeff = G / betabulk * baryon * bulkPi;
        bulk2_coeff = bulkPi / (3.0 * T * betabulk);
        break;
      }
      case 4: // Jonah
      {
        bulk_mod = lambda;
        T_mod = T;
        alphaB_mod = 0.0;

        bulk0_coeff = delta_z  -  3.0 * delta_lambda;
        bulk1_coeff = delta_lambda / T;
        break;
      }
      default:
      {
        printf("Error: set df_mode = (3,3) in parameters.dat\n");
      }
    }

    double Axx = 1.0  +  pixx_LRF * shear_mod  +  bulk_mod;
    double Axy = pixy_LRF * shear_mod;
    double Axz = pixz_LRF * shear_mod;
    double Ayx = Axy;
    double Ayy = 1.0  +  piyy_LRF * shear_mod  +  bulk_mod;
    double Ayz = piyz_LRF * shear_mod;
    double Azx = Axz;
    double Azy = Ayz;
    double Azz = 1.0  +  pizz_LRF * shear_mod  +  bulk_mod;

    double detA = Axx * (Ayy * Azz  -  Ayz * Ayz)  -  Axy * (Axy * Azz  -  Ayz * Axz)  +  Axz * (Axy * Ayz  -  Ayy * Axz);
    double detA_bulk_two_thirds = (1.0 + bulk_mod) * (1.0 + bulk_mod);

    double neq_fact = T * T * T / two_pi2_hbarC3;
    double dn_fact = bulkPi / betabulk;
    double J20_fact = T * neq_fact;
    double N10_fact = neq_fact;
    double nmod_fact = T_mod * T_mod * T_mod / two_pi2_hbarC3;

    // determine if feqmod breaks down
    bool feqmod_breaks_down = does_feqmod_breakdown(MASS_PION0, T, F, bulkPi, betabulk, detA, DETA_MIN, z, laguerre, DF_MODE, 0, T, F, betabulk);


    // I need to use my old method (no matrix solver iterations)


    // uniformly rescale eta space by detA if modified momentum space elements are shrunk
    // this rescales the dsigma components orthogonal to the eta direction (only works for 2+1d, y = 0)
    // for integrating modified distribution with narrow (y-eta) distributions
    double eta_scale = 1.0;
    if(detA > DETA_MIN && DIMENSION == 2)
    {
      eta_scale = detA / detA_bulk_two_thirds;
    }

    // compute renormalization factor
    double renorm = 1.0;

    if(INCLUDE_BULK_DELTAF)
    {
      if(DF_MODE == 3)
      {
        double mbar = mass / T;
        double mbar_mod = mass / T_mod;

        // maybe compute these individually on the device
        double neq = neq_fact * degeneracy * GaussThermal(neq_int, pbar_root1, pbar_weight1, pbar_pts, mbar, alphaB, baryon, sign);

        double N10 = baryon * N10_fact * degeneracy * GaussThermal(J10_int, pbar_root1, pbar_weight1, pbar_pts, mbar, alphaB, baryon, sign);

        double J20 = J20_fact * degeneracy * GaussThermal(J20_int, pbar_root2, pbar_weight2, pbar_pts, mbar, alphaB, baryon, sign);

        double n_linear = neq  +  dn_fact * (neq  +  N10 * G  +  J20 * F / T / T);

        double n_mod = nmod_fact * degeneracy * GaussThermal(neq_int, pbar_root1, pbar_weight1, pbar_pts, mbar_mod, alphaB_mod, baryon, sign);

        renorm = n_linear / n_mod;
      }
      else if(DF_MODE == 4)
      {
        renorm = z;
      }
    }

    // rescale normalization factor
    if(DIMENSION == 2)
    {
      renorm /= detA_bulk_two_thirds;
    }
    else if(DIMENSION == 3)
    {
      renorm /= detA;
    }



    // loop over momentum
    for(long ipT = 0; ipT < pT_tab_length; ipT++)
    {
      double pT = pT_d[ipT];

      double mT = sqrt(mass_squared  +  pT * pT);
      double mT_over_tau = mT / tau;

      for(long iphip = 0; iphip < phi_tab_length; iphip++)
      {
        double px = pT * trig_d[iphip];
        double py = pT * trig_d[iphip + phi_tab_length];

        double px_dax = px * dax;   // useful expressions
        double py_day = py * day;

        double px_ux = px * ux;
        double py_uy = py * uy;

        double pixx_px_px = pixx * px * px;
        double piyy_py_py = piyy * py * py;
        double pitx_px = pitx * px;
        double pity_py = pity * py;
        double pixy_px_py = pixy * px * py;
        double tau2_pixn_px = tau2_pixn * px;
        double tau2_piyn_py = tau2_piyn * py;

        for(long iy = 0; iy < y_tab_length; iy++)
        {
          double y = y_d[iy];

          // momentum_length index
          long iP = ipT  +  pT_tab_length * (iphip  +  phi_tab_length * iy);

          if(positive_time_volume)
          {
            double eta_integral = 0.0;

            // sum over eta
            for(long ieta = 0; ieta < eta_tab_length; ieta++)
            {
              double eta = etaValues_d[ieta];
              double eta_weight = etaWeights_d[ieta];

              bool feqmod_breaks_down_narrow = false;

              if(DIMENSION == 3 && !feqmod_breaks_down)
              {
                if(detA < 0.01 && fabs(y - eta) < detA)
                {
                  feqmod_breaks_down_narrow = true;
                }
              }

              double sinhyeta = sinh(y - eta);
              double coshyeta = sqrt(1.0  +  sinhyeta * sinhyeta);

              double pt = mT * coshyeta;           // p^tau
              double pn = mT_over_tau * sinhyeta;  // p^eta

              double pdotdsigma = pt * dat  +  px_dax  +  py_day  +  pn * dan;

              if(OUTFLOW && pdotdsigma <= 0.0) continue;  // enforce outflow



              double f;

              // calculate feqmod
              if(feqmod_breaks_down || feqmod_breaks_down_narrow)
              {
                double sinhyeta = sinh(y - eta);
                double coshyeta = sqrt(1.0  +  sinhyeta * sinhyeta);

                double pt = mT * coshyeta;           // p^tau
                double pn = mT_over_tau * sinhyeta;  // p^eta

                double pdotdsigma = pt * dat  +  px_dax  +  py_day  +  pn * dan;

                if(OUTFLOW && pdotdsigma <= 0.0) continue;  // enforce outflow

                double E = pt * ut  -  px_ux  -  py_uy  -  pn * tau2_un;  // u.p
                double feq = 1.0 / (exp(E/T  -  chem) + sign);
                double feqbar = 1.0  -  sign * feq;

                // pi^munu.p_mu.p_nu
                double pimunu_pmu_pnu = pitt * pt * pt  +  pixx_px_px  +  piyy_py_py  +  tau4_pinn * pn * pn
                    + 2.0 * (-(pitx_px + pity_py) * pt  +  pixy_px_py  +  pn * (tau2_pixn_px  +  tau2_piyn_py  -  tau2_pitn * pt));

                double df;

                if(DF_MODE == 3)
                {
                  double df_shear = shear_coeff * pimunu_pmu_pnu / E;
                  double df_bulk = (bulk0_coeff * E  +  bulk1_coeff  +  bulk2_coeff * (E  -  mass_squared / E));

                  df = feqbar * (df_shear + df_bulk);
                }
                else if(DF_MODE == 4)
                {
                  double df_shear = feqbar * shear_coeff * pimunu_pmu_pnu / E;
                  double df_bulk = bulk0_coeff  +  feqbar * bulk1_coeff * (E  -  mass_squared / E);

                  df = df_shear + df_bulk;
                }

                if(REGULATE_DELTAF) df = max(-1.0, min(df, 1.0)); // regulate df

                f = feq * (1.0 + df);

              } // feqmod breaks down
              else
              {
                double sinhyeta_scale = sinh(y - eta_scale * eta);

                double pt = mT * sqrt(1.0  +  sinhyeta_scale * sinhyeta_scale);  // p^\tau (GeV)
                double pn = mT_over_tau *  sinhyeta_scale                        // p^\eta (GeV^2)
                double tau2_pn = tau2 * pn;

                // LRF momentum components pi_LRF = - Xi.p
                double px_LRF = -Xt * pt  +  Xx * px  +  Xy * py  +  tau2_Xn * pn;
                double py_LRF = Yx * px  +  Yy * py;
                double pz_LRF = -Zt * pt  +  tau2_Zn * pn;

                double pLRF[3] = {px_LRF, py_LRF, pz_LRF};

                // this is where I need the old method
                //matrix_multiplication(A_inv, pLRF, pLRF_mod, 3, 3);   // evaluate p_mod = A^-1.p at least once

                double px_LRF_mod = pLRF[0];
                double py_LRF_mod = pLRF[1];
                double pz_LRF_mod = pLRF[2];

                double E_mod = sqrt(mass_squared  +  px_LRF_mod * px_LRF_mod  +  py_LRF_mod * py_LRF_mod  +  pz_LRF_mod * pz_LRF_mod);

                f = fabs(renorm) / (exp(E_mod / T_mod  -  chem_mod) + sign); // feqmod
              }

              eta_integral += eta_weight * pdotdsigma * f;

            } // ieta

            dN_pTdpTdphidy_thread[ithread] = prefactor * eta_integral;
          }
          else
          {
            dN_pTdpTdphidy_thread[ithread] = 0.0;
          }

          // perform reduction over threads in each block:
          int N = blockDim.x;  // number of threads in block (must be power of 2)
          __syncthreads();     // prepare threads for reduction

          while(N != 1)
          {
            N /= 2;

            if(ithread < N) // reduce thread pairs
            {
              dN_pTdpTdphidy_thread[ithread] += dN_pTdpTdphidy_thread[ithread + N];
            }
            __syncthreads();
          }

          // store block's contribution to the spectra
          if(ithread == 0)
          {
            long iP_block = iP  +  blockIdx.x * momentum_length;

            dN_pTdpTdphidy_d_blocks[iP_block] = dN_pTdpTdphidy_thread[0];
          }

        } // iy

      } // iphip

    } // ipT

  } // icell < endFO

} // end function

*/

/*

__global__ void calculate_dN_pTdpTdphidy_feqmod_threadReduction(double* dN_pTdpTdphidy_d_blocks, long endFO, long momentum_length, long pT_tab_length, long phi_tab_length, long y_tab_length, long eta_tab_length, double* pT_d, double* trig_d, double* y_d, double* etaValues_d, double* etaWeights_d, double mass_squared, double sign, double prefactor, double baryon, double *T_d, double *tau_d, double *eta_d, double *ux_d, double *uy_d, double *un_d, double *dat_d, double *dax_d, double *day_d, double *dan_d, double *pixx_d, double *pixy_d, double *pixn_d, double *piyy_d, double *piyn_d, double *bulkPi_d, double *alphaB_d, double *Vx_d, double *Vy_d, double *Vn_d, deltaf_coefficients *df_coeff_d, int INCLUDE_BARYON, int REGULATE_DELTAF, int INCLUDE_SHEAR_DELTAF, int INCLUDE_BULK_DELTAF, int INCLUDE_BARYONDIFF_DELTAF, int DIMENSION, int OUTFLOW, int DF_MODE, double DETA_MIN, double MASS_PION0, Gauss_Laguerre * laguerre)
{

  // contribution of dN_pTdpTdphidy from each thread
  __shared__ double dN_pTdpTdphidy_thread[THREADS_PER_BLOCK];

  long icell = (long)threadIdx.x +  (long)blockDim.x * (long)blockIdx.x;  // thread index in kernel
  int ithread = threadIdx.x;                                              // thread index in block

  dN_pTdpTdphidy_thread[ithread] = 0.0;                                   // initialize thread contribution to zero

  __syncthreads();                                                        // sync threads for each block

  if(icell < endFO)
  {
    /// gauss laguerre roots
    const int pbar_pts = laguerre->points;

    double * pbar_root1 = laguerre->root[1];
    double * pbar_root2 = laguerre->root[2];

    double * pbar_weight1 = laguerre->weight[1];
    double * pbar_weight2 = laguerre->weight[2];

    double A[3][3];

    double tau = tau_d[icell];        // longitudinal proper time
    double tau2 = tau * tau;

    if(DIMENSION == 3)
    {
      etaValues_d[0] = eta_d[icell];  // spacetime rapidity from surface
    }
    double dat = dat_d[icell];        // dsigma_mu
    double dax = dax_d[icell];
    double day = day_d[icell];
    double dan = dan_d[icell];

    double ux = ux_d[icell];          // u^mu
    double uy = uy_d[icell];          // enforce u.u = 1
    double un = un_d[icell];
    double ux2 = ux * ux;             // useful expressions
    double uy2 = uy * uy;
    double uperp == sqrt(ux2 + uy2);
    double utperp = sqrt(1.0 + ux2 + uy2);
    double tau2_un = tau2 * un;
    double ut = sqrt(utperp * utperp  +  tau2_un * un);
    double ut2 = ut * ut;

    bool positive_time_volume = (ut * dat  +  ux * dax  +  uy * day  +  un * dan > 0.0);

    double T = T_d[icell];            // temperature

    double pitt = 0.0;                // pi^munu
    double pitx = 0.0;                // enforce pi.u = 0, Tr(pi) = 0
    double pity = 0.0;
    double pitn = 0.0;
    double pixx = 0.0;
    double pixy = 0.0;
    double pixn = 0.0;
    double piyy = 0.0;
    double piyn = 0.0;
    double pinn = 0.0;

    if(INCLUDE_SHEAR_DELTAF)
    {
      pixx = pixx_d[icell];
      pixy = pixy_d[icell];
      pixn = pixn_d[icell];
      piyy = piyy_d[icell];
      piyn = piyn_d[icell];
      pinn = (pixx * (ux2 - ut2)  +  piyy * (uy2 - ut2)  +  2.0 * (pixy * ux * uy  +  tau2_un * (pixn * ux  +  piyn * uy))) / (tau2 * utperp * utperp);
      pitn = (pixn * ux  +  piyn * uy  +  tau2_un * pinn) / ut;
      pity = (pixy * ux  +  piyy * uy  +  tau2_un * piyn) / ut;
      pitx = (pixx * ux  +  pixy * uy  +  tau2_un * pixn) / ut;
      pitt = (pitx * ux  +  pity * uy  +  tau2_un * pitn) / ut;
    }

    double bulkPi = 0.0;              // bulk pressure

    if(INCLUDE_BULK_DELTAF)
    {
      bulkPi = bulkPi_d[icell];
    }

    double alphaB = 0.0;              // muB / T
    double Vt = 0.0;                  // V^mu
    double Vx = 0.0;                  // enforce orthogonality V.u = 0
    double Vy = 0.0;
    double Vn = 0.0;
    double chem = 0.0;

    if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
    {
      alphaB = alphaB_d[icell];
      chem = baryon * alphaB;
      Vx = Vx_d[icell];
      Vy = Vy_d[icell];
      Vn = Vn_d[icell];
      Vt = (Vx * ux  +  Vy * uy  +  Vn * tau2_un) / ut;
    }

    // need basis vector components
    double sinhL = tau * un / utperp;
    double coshL = ut / utperp;

    double Xt = uperp * coshL;
    double Xn = uperp * sinhL / tau;
    double Zt = sinhL;
    double Zn = coshL / tau;

    double tau2_Xn = tau2 * Xn;
    double tau2_Zn = tau2 * Zn;

    double Xx, Xy;
    double Yx, Yy;

    if(uperp < 1.e-5)
    {
      // stops (ux=0)/(uperp=0) nans for freezeout cells with no transverse flow
      Xx = 1.0;   Yx = 0.0;
      Xy = 0.0;   Yy = 1.0;
    }
    else
    {
        Xx = utperp * ux / uperp;   Yx = - uy / uperp;
        Xy = utperp * uy / uperp;   Yy = ux / uperp;
    }

    // pimunu LRF components
    double pixx_LRF = pitt*Xt*Xt + pixx*Xx*Xx + piyy*Xy*Xy + tau2*tau2*pinn*Xn*Xn
            + 2.0 * (-Xt*(pitx*Xx + pity*Xy) + pixy*Xx*Xy + tau2*Xn*(pixn*Xx + piyn*Xy - pitn*Xt));

    double pixy_LRF = Yx*(-pitx*Xt + pixx*Xx + pixy*Xy + tau2*pixn*Xn) + Yy*(-pity*Xt + pixy*Xx + piyy*Xy + tau2*piyn*Xn);

    double pixz_LRF = Zt*(pitt*Xt - pitx*Xx - pity*Xy - tau2*pitn*Xn) - tau2*Zn*(pitn*Xt - pixn*Xx - piyn*Xy - tau2*pinn*Xn);

    double piyy_LRF = pixx*Yx*Yx + 2.0*pixy*Yx*Yy + piyy*Yy*Yy;

    double piyz_LRF = -Zt*(pitx*Yx + pity*Yy) + tau2*Zn*(pixn*Yx + piyn*Yy);

    double pizz_LRF = - (pixx_LRF + piyy_LRF);

    // get df coefficients
    double F = df_coeff_d->F;
    double G = df_coeff_d->G;
    double betabulk = df_coeff_d->betabulk;
    double baryon_enthalpy_ratio = df_coeff_d->baryon_enthalpy_ratio;
    double betaV = df_coeff_d->betaV;
    double betapi = df_coeff_d->betapi;

    double z = df_coeff_d->z;
    double lambda = df_coeff_d->lambda;
    double delta_z = df_coeff_d->delta_z;
    double delta_lambda = df_coeff_d->delta_lambda;


    double shear_mod = 0.5 / betapi;
    double bulk_mod, T_mod, chem_mod;

    // linearized shear and bulk coefficients
    double shear_coeff = 0.5 / (betapi * T);
    double bulk0_coeff, bulk1_coeff, bulk2_coeff;

    switch(DF_MODE)
    {
      case 3: // Mike
      {
        bulk_mod = bulkPi / (3.0 * betabulk);
        T_mod = T  +  F * bulkPi / betabulk;
        chem_mod = baryon * (alphaB  + G * bulkPi / betabulk);

        bulk0_coeff = F / (T * T * betabulk) * bulkPi;
        bulk1_coeff = G / betabulk * baryon * bulkPi;
        bulk2_coeff = bulkPi / (3.0 * T * betabulk);
        break;
      }
      case 4: // Jonah
      {
        bulk_mod = lambda;
        T_mod = T;
        alphaB_mod = 0.0;

        bulk0_coeff = delta_z  -  3.0 * delta_lambda;
        bulk1_coeff = delta_lambda / T;
        break;
      }
      default:
      {
        printf("Error: set df_mode = (3,3) in parameters.dat\n");
      }
    }

    double Axx = 1.0  +  pixx_LRF * shear_mod  +  bulk_mod;
    double Axy = pixy_LRF * shear_mod;
    double Axz = pixz_LRF * shear_mod;
    double Ayx = Axy;
    double Ayy = 1.0  +  piyy_LRF * shear_mod  +  bulk_mod;
    double Ayz = piyz_LRF * shear_mod;
    double Azx = Axz;
    double Azy = Ayz;
    double Azz = 1.0  +  pizz_LRF * shear_mod  +  bulk_mod;

    double detA = Axx * (Ayy * Azz  -  Ayz * Ayz)  -  Axy * (Axy * Azz  -  Ayz * Axz)  +  Axz * (Axy * Ayz  -  Ayy * Axz);
    double detA_bulk_two_thirds = (1.0 + bulk_mod) * (1.0 + bulk_mod);

    double neq_fact = T * T * T / two_pi2_hbarC3;
    double dn_fact = bulkPi / betabulk;
    double J20_fact = T * neq_fact;
    double N10_fact = neq_fact;
    double nmod_fact = T_mod * T_mod * T_mod / two_pi2_hbarC3;

    // determine if feqmod breaks down
    bool feqmod_breaks_down = does_feqmod_breakdown(MASS_PION0, T, F, bulkPi, betabulk, detA, DETA_MIN, z, laguerre, DF_MODE, 0, T, F, betabulk);


    // I need to use my old method (no matrix solver iterations)


    // uniformly rescale eta space by detA if modified momentum space elements are shrunk
    // this rescales the dsigma components orthogonal to the eta direction (only works for 2+1d, y = 0)
    // for integrating modified distribution with narrow (y-eta) distributions
    double eta_scale = 1.0;
    if(detA > DETA_MIN && DIMENSION == 2)
    {
      eta_scale = detA / detA_bulk_two_thirds;
    }

    // compute renormalization factor
    double renorm = 1.0;

    if(INCLUDE_BULK_DELTAF)
    {
      if(DF_MODE == 3)
      {
        double mbar = mass / T;
        double mbar_mod = mass / T_mod;

        // maybe compute these individually on the device
        double neq = neq_fact * degeneracy * GaussThermal(neq_int, pbar_root1, pbar_weight1, pbar_pts, mbar, alphaB, baryon, sign);

        double N10 = baryon * N10_fact * degeneracy * GaussThermal(J10_int, pbar_root1, pbar_weight1, pbar_pts, mbar, alphaB, baryon, sign);

        double J20 = J20_fact * degeneracy * GaussThermal(J20_int, pbar_root2, pbar_weight2, pbar_pts, mbar, alphaB, baryon, sign);

        double n_linear = neq  +  dn_fact * (neq  +  N10 * G  +  J20 * F / T / T);

        double n_mod = nmod_fact * degeneracy * GaussThermal(neq_int, pbar_root1, pbar_weight1, pbar_pts, mbar_mod, alphaB_mod, baryon, sign);

        renorm = n_linear / n_mod;
      }
      else if(DF_MODE == 4)
      {
        renorm = z;
      }
    }

    // rescale normalization factor
    if(DIMENSION == 2)
    {
      renorm /= detA_bulk_two_thirds;
    }
    else if(DIMENSION == 3)
    {
      renorm /= detA;
    }



    // loop over momentum
    for(long ipT = 0; ipT < pT_tab_length; ipT++)
    {
      double pT = pT_d[ipT];

      double mT = sqrt(mass_squared  +  pT * pT);
      double mT_over_tau = mT / tau;

      for(long iphip = 0; iphip < phi_tab_length; iphip++)
      {
        double px = pT * trig_d[iphip];
        double py = pT * trig_d[iphip + phi_tab_length];

        double px_dax = px * dax;   // useful expressions
        double py_day = py * day;

        double px_ux = px * ux;
        double py_uy = py * uy;

        double pixx_px_px = pixx * px * px;
        double piyy_py_py = piyy * py * py;
        double pitx_px = pitx * px;
        double pity_py = pity * py;
        double pixy_px_py = pixy * px * py;
        double tau2_pixn_px = tau2_pixn * px;
        double tau2_piyn_py = tau2_piyn * py;

        for(long iy = 0; iy < y_tab_length; iy++)
        {
          double y = y_d[iy];

          // momentum_length index
          long iP = ipT  +  pT_tab_length * (iphip  +  phi_tab_length * iy);

          if(positive_time_volume)
          {
            double eta_integral = 0.0;

            // sum over eta
            for(long ieta = 0; ieta < eta_tab_length; ieta++)
            {
              double eta = etaValues_d[ieta];
              double eta_weight = etaWeights_d[ieta];

              bool feqmod_breaks_down_narrow = false;

              if(DIMENSION == 3 && !feqmod_breaks_down)
              {
                if(detA < 0.01 && fabs(y - eta) < detA)
                {
                  feqmod_breaks_down_narrow = true;
                }
              }

              double sinhyeta = sinh(y - eta);
              double coshyeta = sqrt(1.0  +  sinhyeta * sinhyeta);

              double pt = mT * coshyeta;           // p^tau
              double pn = mT_over_tau * sinhyeta;  // p^eta

              double pdotdsigma = pt * dat  +  px_dax  +  py_day  +  pn * dan;

              if(OUTFLOW && pdotdsigma <= 0.0) continue;  // enforce outflow



              double f;

              // calculate feqmod
              if(feqmod_breaks_down || feqmod_breaks_down_narrow)
              {
                double sinhyeta = sinh(y - eta);
                double coshyeta = sqrt(1.0  +  sinhyeta * sinhyeta);

                double pt = mT * coshyeta;           // p^tau
                double pn = mT_over_tau * sinhyeta;  // p^eta

                double pdotdsigma = pt * dat  +  px_dax  +  py_day  +  pn * dan;

                if(OUTFLOW && pdotdsigma <= 0.0) continue;  // enforce outflow

                double E = pt * ut  -  px_ux  -  py_uy  -  pn * tau2_un;  // u.p
                double feq = 1.0 / (exp(E/T  -  chem) + sign);
                double feqbar = 1.0  -  sign * feq;

                // pi^munu.p_mu.p_nu
                double pimunu_pmu_pnu = pitt * pt * pt  +  pixx_px_px  +  piyy_py_py  +  tau4_pinn * pn * pn
                    + 2.0 * (-(pitx_px + pity_py) * pt  +  pixy_px_py  +  pn * (tau2_pixn_px  +  tau2_piyn_py  -  tau2_pitn * pt));

                double df;

                if(DF_MODE == 3)
                {
                  double df_shear = shear_coeff * pimunu_pmu_pnu / E;
                  double df_bulk = (bulk0_coeff * E  +  bulk1_coeff  +  bulk2_coeff * (E  -  mass_squared / E));

                  df = feqbar * (df_shear + df_bulk);
                }
                else if(DF_MODE == 4)
                {
                  double df_shear = feqbar * shear_coeff * pimunu_pmu_pnu / E;
                  double df_bulk = bulk0_coeff  +  feqbar * bulk1_coeff * (E  -  mass_squared / E);

                  df = df_shear + df_bulk;
                }

                if(REGULATE_DELTAF) df = max(-1.0, min(df, 1.0)); // regulate df

                f = feq * (1.0 + df);

              } // feqmod breaks down
              else
              {
                double sinhyeta_scale = sinh(y - eta_scale * eta);

                double pt = mT * sqrt(1.0  +  sinhyeta_scale * sinhyeta_scale);  // p^\tau (GeV)
                double pn = mT_over_tau *  sinhyeta_scale                        // p^\eta (GeV^2)
                double tau2_pn = tau2 * pn;

                // LRF momentum components pi_LRF = - Xi.p
                double px_LRF = -Xt * pt  +  Xx * px  +  Xy * py  +  tau2_Xn * pn;
                double py_LRF = Yx * px  +  Yy * py;
                double pz_LRF = -Zt * pt  +  tau2_Zn * pn;

                double pLRF[3] = {px_LRF, py_LRF, pz_LRF};

                // this is where I need the old method
                //matrix_multiplication(A_inv, pLRF, pLRF_mod, 3, 3);   // evaluate p_mod = A^-1.p at least once

                double px_LRF_mod = pLRF[0];
                double py_LRF_mod = pLRF[1];
                double pz_LRF_mod = pLRF[2];

                double E_mod = sqrt(mass_squared  +  px_LRF_mod * px_LRF_mod  +  py_LRF_mod * py_LRF_mod  +  pz_LRF_mod * pz_LRF_mod);

                f = fabs(renorm) / (exp(E_mod / T_mod  -  chem_mod) + sign); // feqmod
              }

              eta_integral += eta_weight * pdotdsigma * f;

            } // ieta

            dN_pTdpTdphidy_thread[ithread] = prefactor * eta_integral;
          }
          else
          {
            dN_pTdpTdphidy_thread[ithread] = 0.0;
          }

          // perform reduction over threads in each block:
          int N = blockDim.x;  // number of threads in block (must be power of 2)
          __syncthreads();     // prepare threads for reduction

          while(N != 1)
          {
            N /= 2;

            if(ithread < N) // reduce thread pairs
            {
              dN_pTdpTdphidy_thread[ithread] += dN_pTdpTdphidy_thread[ithread + N];
            }
            __syncthreads();
          }

          // store block's contribution to the spectra
          if(ithread == 0)
          {
            long iP_block = iP  +  blockIdx.x * momentum_length;

            dN_pTdpTdphidy_d_blocks[iP_block] = dN_pTdpTdphidy_thread[0];
          }

        } // iy

      } // iphip

    } // ipT

  } // icell < endFO

} // end function

*/


// does a block reduction, where the previous kernel did a thread reduction.
__global__ void calculate_dN_pTdpTdphidy_blockReduction(double *dN_pTdpTdphidy_d, double* dN_pTdpTdphidy_d_blocks, long momentum_length, long blocks_ker1, long ipart, long npart)
{
  long ithread = (long)threadIdx.x  +  (long)blockDim.x * (long)blockIdx.x;

  // each thread is assigned a momentum coordinate
  if(ithread < momentum_length)
  {
    long iS3D = ipart + npart * ithread;

    // sum spectra contributions of the blocks from first kernel
    for(long iblock_ker1 = 0; iblock_ker1 < blocks_ker1; iblock_ker1++)
    {
      dN_pTdpTdphidy_d[iS3D] += dN_pTdpTdphidy_d_blocks[ithread  +  iblock_ker1 * momentum_length];
    }
  }
}

// does a block reduction, where the previous kernel did a thread reduction.
__global__ void calculate_dN_dX_blockReduction(double *dN_dX_d, double* dN_dX_d_blocks, long spacetime_length, long blocks_ker1, long ipart, long npart)
{
  long ithread = (long)threadIdx.x  +  (long)blockDim.x * (long)blockIdx.x;

  // each thread is assigned a spacetime coordinate
  if(ithread < spacetime_length)
  {
    long iX = ipart  +  npart * ithread;

    // sum spacetime contributions of the blocks from first kernel
    for(long iblock_ker1 = 0; iblock_ker1 < blocks_ker1; iblock_ker1++)
    {
      dN_dX_d[iX] += dN_dX_d_blocks[ithread  +  iblock_ker1 * spacetime_length];
    }
  }
}



void EmissionFunctionArray::write_dN_2pipTdpTdy_toFile(long *MCID)
{
  char filename[255] = "";
  printf("Writing thermal dN_2pipTdpTdy to file...\n");

  for(long ipart = 0; ipart < npart; ipart++)
  {
    sprintf(filename, "results/continuous/dN_2pipTdpTdy_%ld.dat", MCID[ipart]);
    ofstream spectra(filename, ios_base::out);

    for(long iy = 0; iy < y_tab_length; iy++)
    {
      double y = 0.0;
      if(DIMENSION == 3) y = y_tab->get(1, iy + 1);

      for(long ipT = 0; ipT < pT_tab_length; ipT++)
      {
        double pT =  pT_tab->get(1, ipT + 1);

        double dN_2pipTdpTdy = 0.0;

        for(long iphip = 0; iphip < phi_tab_length; iphip++)
        {
          double phip_weight = phi_tab->get(2, iphip + 1);

          long iS3D = ipart  +  npart * (iy  +  y_tab_length * (iphip  +  phi_tab_length * ipT));

          dN_2pipTdpTdy += phip_weight * dN_pTdpTdphidy[iS3D] / two_pi;

        }

        spectra << scientific <<  setw(5) << setprecision(8) << y << "\t" << pT << "\t" << dN_2pipTdpTdy << "\n";
      }

      if(iy < y_tab_length - 1) spectra << "\n";
    }

    spectra.close();
  }
}

void EmissionFunctionArray::write_vn_toFile(long *MCID)
{
  char filename[255] = "";
  printf("Writing thermal vn to file...\n");

  const complex<double> I(0.0,1.0);   // imaginary i

  const int k_max = 7;                // v_n = {v_1, ..., v_7}

  for(long ipart = 0; ipart < npart; ipart++)
  {
    sprintf(filename, "results/continuous/vn_%ld.dat", MCID[ipart]);
    ofstream vn_File(filename, ios_base::out);

    for(long iy = 0; iy < y_tab_length; iy++)
    {
      double y = 0.0;
      if(DIMENSION == 3) y = y_tab->get(1, iy + 1);

      for(long ipT = 0; ipT < pT_tab_length; ipT++)
      {
        double pT = pT_tab->get(1, ipT + 1);

        double Vn_real_numerator[k_max];
        double Vn_imag_numerator[k_max];

        for(int k = 0; k < k_max; k++)
        {
          Vn_real_numerator[k] = 0.0;
          Vn_imag_numerator[k] = 0.0;
        }

        double vn_denominator = 0.0;

        for(long iphip = 0; iphip < phi_tab_length; iphip++)
        {
          double phip = phi_tab->get(1, iphip + 1);
          double phip_weight = phi_tab->get(2, iphip + 1);

          long iS3D = ipart  +  npart * (iy  +  y_tab_length * (iphip  +  phi_tab_length * ipT));

          for(int k = 0; k < k_max; k++)
          {
            Vn_real_numerator[k] += cos(((double)k + 1.0) * phip) * phip_weight * dN_pTdpTdphidy[iS3D];
            Vn_imag_numerator[k] += sin(((double)k + 1.0) * phip) * phip_weight * dN_pTdpTdphidy[iS3D];
          }

          vn_denominator += phip_weight * dN_pTdpTdphidy[iS3D];
        } 

        vn_File << scientific <<  setw(5) << setprecision(8) << y << "\t" << pT;

        for(long k = 0; k < k_max; k++)
        {
          double vn = abs(Vn_real_numerator[k]  +  I * Vn_imag_numerator[k]) / vn_denominator;

          if(vn_denominator < 1.e-15) vn = 0.0;

          vn_File << "\t" << vn;
        }

        vn_File << "\n";
      }

       if(iy < y_tab_length - 1) vn_File << "\n";
    } 

    vn_File.close();
  }
}


void EmissionFunctionArray::write_dN_dphidy_toFile(long *MCID)
{
  char filename[255] = "";
  printf("Writing thermal dN_dphidy to file...\n");

  for(long ipart  = 0; ipart < npart; ipart++)
  {
    sprintf(filename, "results/continuous/dN_dphipdy_%ld.dat", MCID[ipart]);
    ofstream spectra(filename, ios_base::app);

    for(long iy = 0; iy < y_tab_length; iy++)
    {
      double y = 0.0;
      if(DIMENSION == 3) y = y_tab->get(1, iy + 1);

      for(long iphip = 0; iphip < phi_tab_length; iphip++)
      {
        double phip = phi_tab->get(1,iphip + 1);

        double dN_dphipdy = 0.0;

        for(long ipT = 0; ipT < pT_tab_length; ipT++)
        {
          double pT_weight = pT_tab->get(2, ipT + 1);

          long iS3D = ipart  +  npart * (iy  +  y_tab_length * (iphip  +  phi_tab_length * ipT));

          dN_dphipdy += pT_weight * dN_pTdpTdphidy[iS3D];
        }

        spectra << scientific <<  setw(5) << setprecision(8) << y << "\t" << phip << "\t" << dN_dphipdy << "\n";
      }

      if(iy < y_tab_length - 1) spectra << "\n";
    }

    spectra.close();
  }
}



void EmissionFunctionArray::write_dN_dy_toFile(long *MCID)
{
  char filename[255] = "";
  printf("Writing thermal dN_dy to file...\n");

  for(long ipart  = 0; ipart < npart; ipart++)
  {
    sprintf(filename, "results/continuous/dN_dy_%ld.dat", MCID[ipart]);
    ofstream spectra(filename, ios_base::out);

    for(long iy = 0; iy < y_tab_length; iy++)
    {
      double y = 0.0;
      if(DIMENSION == 3) y = y_tab->get(1, iy + 1);

      double dN_dy = 0.0;

      for(long iphip = 0; iphip < phi_tab_length; iphip++)
      {
        double phip_weight = phi_tab->get(2, iphip + 1);

        for(long ipT = 0; ipT < pT_tab_length; ipT++)
        {
          double pT_weight = pT_tab->get(2, ipT + 1);

          long iS3D = ipart  +  npart * (ipT  +  pT_tab_length * (iphip  +  phi_tab_length * iy));

          dN_dy += phip_weight * pT_weight * dN_pTdpTdphidy[iS3D];

        } 
      }

      spectra << setw(5) << setprecision(8) << y << "\t" << dN_dy << "\n";
    } 

    spectra.close();
  }
}

void EmissionFunctionArray::write_dN_taudtaudeta_toFile(long *MCID)
{
  char filename[255] = "";
  printf("Writing thermal dN_taudtaudeta to file...\n");

  for(long ipart = 0; ipart < npart; ipart++)
  {
    sprintf(filename, "results/continuous/dN_taudtaudeta_%ld.dat", MCID[ipart]);
    ofstream dN_taudtaudeta(filename, ios_base::out);

    for(long ieta = 0; ieta < eta_bins; ieta++)
    {
      double eta_mid = 0.0;

      if(DIMENSION == 3) eta_mid = eta_min  +  eta_width * (ieta + 0.5);

      for(long itau = 0; itau < tau_bins; itau++)
      {
        double tau_mid = tau_min  +  tau_width * (itau + 0.5);

        long iX = ipart  +  npart * (ieta  +  eta_bins * itau);

        dN_taudtaudeta << setprecision(6) << scientific << eta_mid << "\t" << tau_mid << "\t" << dN_dX[iX] / (tau_mid * tau_width * eta_width) << "\n";
      }

      if(ieta < eta_bins - 1) dN_taudtaudeta << "\n";
    } 

    dN_taudtaudeta.close();
  }
}

void EmissionFunctionArray::write_dN_2pirdrdeta_toFile(long *MCID)
{
  char filename[255] = "";
  printf("Writing thermal dN_2pirdrdeta to file...\n");

  for(long ipart = 0; ipart < npart; ipart++)
  {
    sprintf(filename, "results/continuous/dN_2pirdrdeta_%ld.dat", MCID[ipart]);
    ofstream dN_2pirdrdeta(filename, ios_base::out);

    for(long ieta = 0; ieta < eta_bins; ieta++)
    {
      double eta_mid = 0.0;

      if(DIMENSION == 3) eta_mid = eta_min  +  eta_width * (ieta + 0.5);

      for(long ir = 0; ir < r_bins; ir++)
      {
        double r_mid = r_min  +  r_width * (ir + 0.5);

        long iX = ipart  +  npart * (ieta  +  eta_bins * (ir + tau_bins));

        dN_2pirdrdeta << setprecision(6) << scientific << eta_mid << "\t" << r_mid << "\t" << dN_dX[iX] / (two_pi * r_mid * r_width * eta_width) << "\n";
      }

      if(ieta < eta_bins - 1) dN_2pirdrdeta << "\n";
    } 

    dN_2pirdrdeta.close();
  }
}

void EmissionFunctionArray::write_dN_dphideta_toFile(long *MCID)
{
  char filename[255] = "";
  printf("Writing thermal dN_dphideta to file...\n");

  for(long ipart = 0; ipart < npart; ipart++)
  {
    sprintf(filename, "results/continuous/dN_dphideta_%ld.dat", MCID[ipart]);
    ofstream dN_dphideta(filename, ios_base::out);

    for(long ieta = 0; ieta < eta_bins; ieta++)
    {
      double eta_mid = 0.0;

      if(DIMENSION == 3) eta_mid = eta_min  +  eta_width * (ieta + 0.5);

      for(long iphi = 0; iphi < phi_bins; iphi++)
      {
        double phi_mid = phi_width * (iphi + 0.5);

        long iX = ipart  +  npart * (ieta  +  eta_bins * (iphi + tau_bins + r_bins));

        dN_dphideta << setprecision(6) << scientific << eta_mid << "\t" << phi_mid << "\t" << dN_dX[iX] / (phi_width * eta_width) << "\n";
      }

      if(ieta < eta_bins - 1) dN_dphideta << "\n";
    } 

    dN_dphideta.close();
  }
}




void EmissionFunctionArray::calculate_spectra()
{
  cout << "calculate_spectra() has started... " << endl;
  Stopwatch sw;
  sw.tic();

  hipDeviceSynchronize();    // test synchronize device
  hipError_t err;            // errors

  err = hipGetLastError();
  if(err != hipSuccess)
  {
    printf("Error at very beginning: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }

  double h3 = pow(2.0 * M_PI * hbarC, 3);

  long chunks = (FO_length + FO_chunk - 1) / FO_chunk;                    // number of chunks
  long partial = FO_length % FO_chunk;                                    // remainder cells

  long blocks_ker1 = (FO_chunk + threadsPerBlock - 1) / threadsPerBlock;  // number of blocks in kernel 1 (thread reduction)
  long blocks_ker2;                                                       // number of blocks in kernel 2 (block reduction)
  long block_length;

  if(OPERATION == 0)  // spacetime distribution
  {
    blocks_ker2 = (spacetime_length + threadsPerBlock - 1) / threadsPerBlock; 
    block_length = blocks_ker1 * spacetime_length;
  } 
  else if(OPERATION == 1) // spectra
  {
    blocks_ker2 = (momentum_length + threadsPerBlock - 1) / threadsPerBlock;
    block_length = blocks_ker1 * momentum_length;
  }


  cout << "________________________|______________" << endl;
  cout << "chosen_particles        |\t" << npart << endl;
  cout << "pT_tab_length           |\t" << pT_tab_length << endl;
  cout << "phi_tab_length          |\t" << phi_tab_length << endl;
  cout << "y_tab_length            |\t" << y_tab_length << endl;
  cout << "eta_tab_length          |\t" << eta_tab_length << endl;
  cout << "momentum length         |\t" << momentum_length << endl;
  cout << "total spectra length    |\t" << spectra_length << endl;
  cout << "tau_bins                |\t" << tau_bins << endl;
  cout << "r_bins                  |\t" << r_bins << endl;
  cout << "phi_bins                |\t" << phi_bins << endl;
  cout << "eta_bins                |\t" << eta_bins << endl;
  cout << "spacetime length        |\t" << spacetime_length << endl;
  cout << "total spacetime length  |\t" << X_length << endl;  
  cout << "block length            |\t" << block_length << endl;
  cout << "freezeout cells         |\t" << FO_length << endl;
  cout << "chunk size              |\t" << FO_chunk << endl;
  cout << "number of chunks        |\t" << chunks << endl;
  cout << "remainder cells         |\t" << partial << endl;
  cout << "blocks in first kernel  |\t" << blocks_ker1 << endl;
  cout << "blocks in second kernel |\t" << blocks_ker2 << endl;
  cout << "threads per block       |\t" << threadsPerBlock << endl;


  cout << "Declaring and filling host arrays with particle and freezeout surface info" << endl;

  // particle info
  particle_info *particle;
  double *Mass = (double*)calloc(npart, sizeof(double));
  double *Sign = (double*)calloc(npart, sizeof(double));
  double *Degen = (double*)calloc(npart, sizeof(double));
  double *Baryon = (double*)calloc(npart, sizeof(double));
  long *MCID = (long*)calloc(npart, sizeof(long));

  for(long ipart = 0; ipart < npart; ipart++)
  {
    long pdg_index = chosen_particles_table[ipart];
    particle = &particles[pdg_index];

    Mass[ipart] = particle->mass;
    Sign[ipart] = particle->sign;
    Degen[ipart] = particle->gspin;
    Baryon[ipart] = particle->baryon;
    MCID[ipart] = particle->mc_id;
  }


  // freezeout surface info
  FO_surf *surf;
  double *T, *alphaB;                         // thermodynamic properties
  double *tau, *x, *y, *eta;                  // position
  double *ux, *uy, *un;                       // u^mu
  double *dat, *dax, *day, *dan;              // dsigma_mu
  double *pixx, *pixy, *pixn, *piyy, *piyn;   // pi^munu
  double *bulkPi;                             // bulk pressure
  double *Vx, *Vy, *Vn;                       // V^mu

  deltaf_coefficients *df_coeff;

  // callocate memory
  T = (double*)calloc(FO_chunk, sizeof(double));

  tau = (double*)calloc(FO_chunk, sizeof(double));
  x = (double*)calloc(FO_chunk, sizeof(double));
  y = (double*)calloc(FO_chunk, sizeof(double));
  if(DIMENSION == 3)
  {
    eta = (double*)calloc(FO_chunk, sizeof(double));
  }

  ux = (double*)calloc(FO_chunk, sizeof(double));
  uy = (double*)calloc(FO_chunk, sizeof(double));
  un = (double*)calloc(FO_chunk, sizeof(double));

  dat = (double*)calloc(FO_chunk, sizeof(double));
  dax = (double*)calloc(FO_chunk, sizeof(double));
  day = (double*)calloc(FO_chunk, sizeof(double));
  dan = (double*)calloc(FO_chunk, sizeof(double));

  if(INCLUDE_SHEAR_DELTAF)
  {
    pixx = (double*)calloc(FO_chunk, sizeof(double));
    pixy = (double*)calloc(FO_chunk, sizeof(double));
    pixn = (double*)calloc(FO_chunk, sizeof(double));
    piyy = (double*)calloc(FO_chunk, sizeof(double));
    piyn = (double*)calloc(FO_chunk, sizeof(double));
  }

  if(INCLUDE_BULK_DELTAF)
  {
    bulkPi = (double*)calloc(FO_chunk, sizeof(double));
  }

  if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
  {
    alphaB = (double*)calloc(FO_chunk, sizeof(double));  // muB / T
    Vx = (double*)calloc(FO_chunk, sizeof(double));
    Vy = (double*)calloc(FO_chunk, sizeof(double));
    Vn = (double*)calloc(FO_chunk, sizeof(double));
  }

  df_coeff = (deltaf_coefficients*)calloc(FO_chunk, sizeof(deltaf_coefficients));

  // set up momentum tables to pass to GPU
  double *pT = (double*)calloc(pT_tab_length, sizeof(double));
  double *trig = (double*)calloc(2 * phi_tab_length, sizeof(double));  // {cos(phip), sin(phip)}
  double *yp = (double*)calloc(y_tab_length, sizeof(double));
  double *etaValues = (double*)calloc(eta_tab_length, sizeof(double));
  double *etaWeights = (double*)calloc(eta_tab_length, sizeof(double));
  double *y_minus_eta = (double*)calloc(y_minus_eta_tab_length, sizeof(double));

  double *pT_weight = (double*)calloc(pT_tab_length, sizeof(double));
  double *phip_weight = (double*)calloc(phi_tab_length, sizeof(double));
  double *y_minus_eta_weight = (double*)calloc(y_minus_eta_tab_length, sizeof(double));

  for(long ipT = 0; ipT < pT_tab_length; ipT++)
  {
    pT[ipT] = pT_tab->get(1, ipT + 1);

    if(OPERATION == 0) 
    {
      pT_weight[ipT] = pT_tab->get(2, ipT + 1);
    }
  }

  for(long iphip = 0; iphip < phi_tab_length; iphip++)
  {
    double phip = phi_tab->get(1, iphip + 1);

    trig[iphip] = cos(phip);
    trig[iphip + phi_tab_length] = sin(phip);

    if(OPERATION == 0)
    {
      phip_weight[iphip] = phi_tab->get(2, iphip + 1);
    }
  }

  if(DIMENSION == 2)
  {
    yp[0] = 0.0;
    for(long ieta = 0; ieta < eta_tab_length; ieta++)
    {
      etaValues[ieta] = eta_tab->get(1, ieta + 1);
      etaWeights[ieta] = eta_tab->get(2, ieta + 1);
    }
  }
  else if(DIMENSION == 3)
  {
    etaValues[0] = 0.0;
    etaWeights[0] = 1.0;
    for(long iy = 0; iy < y_tab_length; iy++)
    {
      yp[iy] = y_tab->get(1, iy + 1);
    }
  }

  for(long iyeta = 0; iyeta < y_minus_eta_tab_length; iyeta++)
  {
    y_minus_eta[iyeta] = eta_tab->get(1, iyeta + 1);
    y_minus_eta_weight[iyeta] = eta_tab->get(2, iyeta + 1);
  }


  cout << "Declaring and allocating device arrays " << endl;

  double *T_d, *alphaB_d;
  double *tau_d, *x_d, *y_d, *eta_d;
  double *ux_d, *uy_d, *un_d;
  double *dat_d, *dax_d, *day_d, *dan_d;
  double *pixx_d, *pixy_d, *pixn_d, *piyy_d, *piyn_d;
  double *bulkPi_d;
  double *Vx_d, *Vy_d, *Vn_d;

  deltaf_coefficients *df_coeff_d;

  double *pT_d, *trig_d, *yp_d, *etaValues_d, *etaWeights_d, *y_minus_eta_d;
  double *pT_weight_d, *phip_weight_d, *y_minus_eta_weight_d;

  double *dN_pTdpTdphidy_d, *dN_dX_d;
  double *dN_pTdpTdphidy_d_blocks, *dN_dX_d_blocks;



  // allocate memory on device
  hipMalloc((void**) &T_d, FO_chunk * sizeof(double));

  hipMalloc((void**) &tau_d, FO_chunk * sizeof(double));
  hipMalloc((void**) &x_d, FO_chunk * sizeof(double));
  hipMalloc((void**) &y_d, FO_chunk * sizeof(double));
  if(DIMENSION == 3)
  {
    hipMalloc((void**) &eta_d, FO_chunk * sizeof(double));
  }

  hipMalloc((void**) &ux_d, FO_chunk * sizeof(double));
  hipMalloc((void**) &uy_d, FO_chunk * sizeof(double));
  hipMalloc((void**) &un_d, FO_chunk * sizeof(double));

  hipMalloc((void**) &dat_d, FO_chunk * sizeof(double));
  hipMalloc((void**) &dax_d, FO_chunk * sizeof(double));
  hipMalloc((void**) &day_d, FO_chunk * sizeof(double));
  hipMalloc((void**) &dan_d, FO_chunk * sizeof(double));

  if(INCLUDE_SHEAR_DELTAF)
  {
    hipMalloc((void**) &pixx_d, FO_chunk * sizeof(double));
    hipMalloc((void**) &pixy_d, FO_chunk * sizeof(double));
    hipMalloc((void**) &pixn_d, FO_chunk * sizeof(double));
    hipMalloc((void**) &piyy_d, FO_chunk * sizeof(double));
    hipMalloc((void**) &piyn_d, FO_chunk * sizeof(double));
  }

  if(INCLUDE_BULK_DELTAF)
  {
    hipMalloc((void**) &bulkPi_d, FO_chunk * sizeof(double));
  }

  if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
  {
    hipMalloc((void**) &alphaB_d, FO_chunk * sizeof(double));
    hipMalloc((void**) &Vx_d, FO_chunk * sizeof(double));
    hipMalloc((void**) &Vy_d, FO_chunk * sizeof(double));
    hipMalloc((void**) &Vn_d, FO_chunk * sizeof(double));
  }

  hipMalloc((void**) &df_coeff_d, FO_chunk * sizeof(deltaf_coefficients));

  hipMalloc((void**) &pT_d,  pT_tab_length * sizeof(double));
  hipMalloc((void**) &trig_d, 2 * phi_tab_length * sizeof(double));
  hipMalloc((void**) &yp_d, y_tab_length * sizeof(double));
  hipMalloc((void**) &etaValues_d, eta_tab_length * sizeof(double));
  hipMalloc((void**) &etaWeights_d, eta_tab_length * sizeof(double));
  hipMalloc((void**) &y_minus_eta_d, y_minus_eta_tab_length * sizeof(double));

  hipMalloc((void**) &pT_weight_d,  pT_tab_length * sizeof(double));
  hipMalloc((void**) &phip_weight_d, phi_tab_length * sizeof(double));
  hipMalloc((void**) &y_minus_eta_weight_d, y_minus_eta_tab_length * sizeof(double));

  if(OPERATION == 0)
  {
    hipMalloc((void**) &dN_dX_d, X_length * sizeof(double));
    hipMalloc((void**) &dN_dX_d_blocks, block_length * sizeof(double));
  }
  else if(OPERATION == 1)
  {
    hipMalloc((void**) &dN_pTdpTdphidy_d, spectra_length * sizeof(double));
    hipMalloc((void**) &dN_pTdpTdphidy_d_blocks, block_length * sizeof(double));
  }
  

  err = hipGetLastError();
  if(err != hipSuccess)
  {
    printf("Error in device memory allocation: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  printf("Copying momentum tables from host to device...\n");
  hipMemcpy(pT_d, pT, pT_tab_length * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(trig_d, trig, 2 * phi_tab_length * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(yp_d, yp, y_tab_length * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(etaValues_d, etaValues, eta_tab_length * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(etaWeights_d, etaWeights, eta_tab_length * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y_minus_eta_d, y_minus_eta, y_minus_eta_tab_length * sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(pT_weight_d, pT_weight, pT_tab_length * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(phip_weight_d, phip_weight, phi_tab_length * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y_minus_eta_weight_d, y_minus_eta_weight, y_minus_eta_tab_length * sizeof(double), hipMemcpyHostToDevice);

  err = hipGetLastError();
  if(err != hipSuccess)
  {
    printf("Error in memory copy from host to device: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  if(OPERATION == 0)
  {
    hipMemset(dN_dX_d, 0.0, X_length * sizeof(double));
  }
  else
  {
    hipMemset(dN_pTdpTdphidy_d, 0.0, spectra_length * sizeof(double));
  }

  err = hipGetLastError();
  if(err != hipSuccess)
  {
    printf("Error in device memory set: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }


  printf("\n");
  

  for(long n = 0; n < chunks; n++)
  {
    long endFO = FO_chunk;

    if((n == chunks - 1) && partial > 0) endFO = partial;

    // get freezeout surface info for the chunk
    for(long icell = 0; icell < endFO; icell++)
    {
      long icell_glb = icell  +  n * FO_chunk;   // global cell index

      surf = &surf_ptr[icell_glb];

      T[icell] = surf->T;
   
      tau[icell] = surf->tau;
      x[icell] = surf->x;
      y[icell] = surf->y;
      if(DIMENSION == 3)
      {
        eta[icell] = surf->eta;
      }

      ux[icell] = surf->ux;
      uy[icell] = surf->uy;
      un[icell] = surf->un;

      dat[icell] = surf->dat;
      dax[icell] = surf->dax;
      day[icell] = surf->day;
      dan[icell] = surf->dan;

      if(INCLUDE_SHEAR_DELTAF)
      {
        pixx[icell] = surf->pixx;
        pixy[icell] = surf->pixy;
        pixn[icell] = surf->pixn;
        piyy[icell] = surf->piyy;
        piyn[icell] = surf->piyn;
      }

      if(INCLUDE_BULK_DELTAF)
      {
        bulkPi[icell] = surf->bulkPi;
      }

      if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
      {
        alphaB[icell] = (surf->muB) / (surf->T);
        Vx[icell] = surf->Vx;
        Vy[icell] = surf->Vy;
        Vn[icell] = surf->Vn;
      }

      // evaluate the df coefficients
      double T_FO = T[icell];
      double P = surf->P;
      double E = surf->E;
      double muB = 0.0;
      double nB = 0.0;
      if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
      {
        muB = T_FO * alphaB[icell];
        nB = surf->nB;
      }
      double bulkPi_FO = 0.0;
      if(INCLUDE_BULK_DELTAF)
      {
        bulkPi_FO = bulkPi[icell];
      }

      df_coeff[icell] = df_data->evaluate_df_coefficients(T_FO, muB, E, P, nB, bulkPi_FO);

    } // icell


    // copy memory from host to device
    hipMemcpy(T_d, T, FO_chunk * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(tau_d, tau, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x_d, x, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    if(DIMENSION == 3)
    {
      hipMemcpy(eta_d, eta, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    }

    hipMemcpy(ux_d, ux, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(uy_d, uy, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(un_d, un, FO_chunk * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(dat_d, dat, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dax_d, dax, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(day_d, day, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dan_d, dan, FO_chunk * sizeof(double), hipMemcpyHostToDevice);

    if(INCLUDE_SHEAR_DELTAF)
    {
      hipMemcpy(pixx_d, pixx, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(pixy_d, pixy, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(pixn_d, pixn, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(piyy_d, piyy, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(piyn_d, piyn, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    }

    if(INCLUDE_BULK_DELTAF)
    {
      hipMemcpy(bulkPi_d, bulkPi, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    }

    if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
    {
      hipMemcpy(alphaB_d, alphaB, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(Vx_d, Vx, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(Vy_d, Vy, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(Vn_d, Vn, FO_chunk * sizeof(double), hipMemcpyHostToDevice);
    }

    hipMemcpy(df_coeff_d, df_coeff, FO_chunk * sizeof(deltaf_coefficients), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if(err != hipSuccess)
    {
      printf("Error in memory copy from host to device: %s\n", hipGetErrorString(err));
      err = hipSuccess;
    }

    // loop over particles
    for(int ipart = 0; ipart < npart; ipart++)
    {
      // reset block spectra to zero
      if(OPERATION == 0)
      {
        hipMemset(dN_dX_d_blocks, 0.0, block_length * sizeof(double));
      }
      else if(OPERATION == 1)
      {
        hipMemset(dN_pTdpTdphidy_d_blocks, 0.0, block_length * sizeof(double));
      }

      double mass = Mass[ipart];
      double sign = Sign[ipart];
      double degen = Degen[ipart];
      double baryon = Baryon[ipart];

      double mass_squared = mass * mass;
      double prefactor = degen / h3;

      switch(DF_MODE)
      {
        case 1:
        case 2:
        {
          // launch thread reduction kernel
          if(OPERATION == 0)
          {
            hipDeviceSynchronize();
            calculate_dN_dX_threadReduction<<<blocks_ker1, threadsPerBlock>>>(dN_dX_d_blocks, endFO, tau_bins, r_bins, phi_bins, eta_bins, tau_min, r_min, eta_min, tau_width, r_width, phi_width, eta_width, pT_tab_length, phi_tab_length, y_minus_eta_tab_length, pT_d, pT_weight_d, trig_d, phip_weight_d, y_minus_eta_d, y_minus_eta_weight_d, mass_squared, sign, prefactor, baryon, T_d, tau_d, x_d, y_d, eta_d, ux_d, uy_d, un_d, dat_d, dax_d, day_d, dan_d, pixx_d, pixy_d, pixn_d, piyy_d, piyn_d, bulkPi_d, alphaB_d, Vx_d, Vy_d, Vn_d, df_coeff_d, INCLUDE_BARYON, REGULATE_DELTAF, INCLUDE_SHEAR_DELTAF, INCLUDE_BULK_DELTAF, INCLUDE_BARYONDIFF_DELTAF, DIMENSION, OUTFLOW, DF_MODE);
            hipDeviceSynchronize();
          }
          else if(OPERATION == 1)
          {
            hipDeviceSynchronize();
            calculate_dN_pTdpTdphidy_threadReduction<<<blocks_ker1, threadsPerBlock, sizeof(double)*threadsPerBlock>>>(dN_pTdpTdphidy_d_blocks, endFO, momentum_length, pT_tab_length, phi_tab_length, y_tab_length, eta_tab_length, pT_d, trig_d, yp_d, etaValues_d, etaWeights_d, mass_squared, sign, prefactor, baryon, T_d, tau_d, eta_d, ux_d, uy_d, un_d, dat_d, dax_d, day_d, dan_d, pixx_d, pixy_d, pixn_d, piyy_d, piyn_d, bulkPi_d, alphaB_d, Vx_d, Vy_d, Vn_d, df_coeff_d, INCLUDE_BARYON, REGULATE_DELTAF, INCLUDE_SHEAR_DELTAF, INCLUDE_BULK_DELTAF, INCLUDE_BARYONDIFF_DELTAF, DIMENSION, OUTFLOW, DF_MODE);
            hipDeviceSynchronize();
          }
          
          break;
        }
        case 3:
        case 4:
        {
          // launch thread reduction kernel
          //hipDeviceSynchronize();
          //calculate_dN_pTdpTdphidy_feqmod_threadReduction<<<blocks_ker1, threadsPerBlock>>>(dN_pTdpTdphidy_d_blocks, endFO, momentum_length, pT_tab_length, phi_tab_length, y_tab_length, eta_tab_length, pT_d, trig_d, yp_d, etaValues_d, etaWeights_d, mass_squared, sign, prefactor, baryon, T_d, P_d, E_d, tau_d, eta_d, ux_d, uy_d, un_d, dat_d, dax_d, day_d, dan_d, pixx_d, pixy_d, pixn_d, piyy_d, piyn_d, bulkPi_d, alphaB_d, nB_d, Vx_d, Vy_d, Vn_d, df_coeff_d, INCLUDE_BARYON, REGULATE_DELTAF, INCLUDE_SHEAR_DELTAF, INCLUDE_BULK_DELTAF, INCLUDE_BARYONDIFF_DELTAF, DIMENSION, OUTFLOW, DF_MODE);
          //hipDeviceSynchronize();

          break;
        }
      }

      err = hipGetLastError();
      if(err != hipSuccess)
      {
        printf("Error in thread reduction kernel: %s\n", hipGetErrorString(err));
        err = hipSuccess;
      }

      // launch block reduction kernel
      if(OPERATION == 0)
      {
        hipDeviceSynchronize();
        calculate_dN_dX_blockReduction<<<blocks_ker2, threadsPerBlock>>>(dN_dX_d, dN_dX_d_blocks, spacetime_length, blocks_ker1, ipart, npart);
        hipDeviceSynchronize();
      }
      else if(OPERATION == 1)
      {
        hipDeviceSynchronize();
        calculate_dN_pTdpTdphidy_blockReduction<<<blocks_ker2, threadsPerBlock>>>(dN_pTdpTdphidy_d, dN_pTdpTdphidy_d_blocks, momentum_length, blocks_ker1, ipart, npart);
        hipDeviceSynchronize();
      }
    
      err = hipGetLastError();
      if(err != hipSuccess)
      {
        printf("Error in block reduction kernel: %s\n", hipGetErrorString(err));
        err = hipSuccess;
      }

    } // ipart

    printf("Progress: finished chunk %ld of %ld", n + 1, chunks);
    printf("\n");

  } // loop over chunks


  // copy results from device to host and write to file
  if(OPERATION == 0)
  {
    hipMemcpy(dN_dX, dN_dX_d, X_length * sizeof(double), hipMemcpyDeviceToHost);

    write_dN_taudtaudeta_toFile(MCID);
    write_dN_2pirdrdeta_toFile(MCID);
    write_dN_dphideta_toFile(MCID);

    free(dN_dX);
  }
  else if(OPERATION == 1)
  {
    hipMemcpy(dN_pTdpTdphidy, dN_pTdpTdphidy_d, spectra_length * sizeof(double), hipMemcpyDeviceToHost);

    write_dN_2pipTdpTdy_toFile(MCID);
    write_vn_toFile(MCID);
    //write_dN_dphidy_toFile(MCID);
    //write_dN_dy_toFile(MCID);

    free(dN_pTdpTdphidy);
  }
 
  cout << "Deallocating host and device memory" << endl;  
  free(chosen_particles_table);

  free(Mass);
  free(Sign);
  free(Degen);
  free(Baryon);
  free(MCID);

  free(T);

  free(tau);
  free(x);
  free(y);
  if(DIMENSION == 3) free(eta);

  free(ux);
  free(uy);
  free(un);

  free(dat);
  free(dax);
  free(day);
  free(dan);

  if(INCLUDE_SHEAR_DELTAF)
  {
    free(pixx);
    free(pixy);
    free(pixn);
    free(piyy);
    free(piyn);
  }

  if(INCLUDE_BULK_DELTAF)
  {
    free(bulkPi);
  }


  if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
  {
    free(alphaB);
    free(Vx);
    free(Vy);
    free(Vn);
  }

  free(pT);
  free(trig);
  free(yp);
  free(etaValues);
  free(etaWeights);
  free(y_minus_eta);
  free(pT_weight);
  free(phip_weight);
  free(y_minus_eta_weight);

  free(df_coeff);

  // hipFree = deallocate memory on device
  hipFree(T_d);

  hipFree(x_d);
  hipFree(y_d);
  hipFree(tau_d);
  if(DIMENSION == 3)
  {
    hipFree(eta_d);
  }

  hipFree(ux_d);
  hipFree(uy_d);
  hipFree(un_d);

  hipFree(dat_d);
  hipFree(dax_d);
  hipFree(day_d);
  hipFree(dan_d);

  if(INCLUDE_SHEAR_DELTAF)
  {
    hipFree(pixx_d);
    hipFree(pixy_d);
    hipFree(pixn_d);
    hipFree(piyy_d);
    hipFree(piyn_d);
  }

  if(INCLUDE_BULK_DELTAF)
  {
    hipFree(bulkPi_d);
  }

  if(INCLUDE_BARYON && INCLUDE_BARYONDIFF_DELTAF)
  {
    hipFree(alphaB_d);
    hipFree(Vx_d);
    hipFree(Vy_d);
    hipFree(Vn_d);
  }

  hipFree(df_coeff_d);

  hipFree(pT_d);
  hipFree(trig_d);
  hipFree(yp_d);
  hipFree(etaValues_d);
  hipFree(etaWeights_d);

  if(OPERATION == 0)
  {
    hipFree(dN_dX_d);
    hipFree(dN_dX_d_blocks);
  }
  else if(OPERATION == 1)
  {
    hipFree(dN_pTdpTdphidy_d);
    hipFree(dN_pTdpTdphidy_d_blocks);
  }

  sw.toc();
  cout << "\nKernel launches took " << sw.takeTime() << " seconds.\n" << endl;
}


